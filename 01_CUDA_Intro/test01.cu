
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>


/*
Compile:   nvcc test01.cu -o test01
Run:       ./test01
Benchmark: nvprof ./test01
*/



__global__
void add(int n, float *x, float *y){
	int index = threadIdx.x;
	int stride = blockDim.x;
	for(int i=index; i < n; i+=stride)
		y[i] = x[i] + y[i];
}


int main(void){
	int N = 1<<31;
	float *x, *y;
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for(int i=0 ; i< N; ++i){
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	add<<<1, 1024>>>(N, x, y);

	hipDeviceSynchronize();

	float maxError = 0.0f;
	for(int i=0; i < N; ++i)
		maxError = fmax(maxError, fabs(y[i]-3.0f));

	std::cout << "Max error: " << maxError << "\n";

	hipFree(x);
	hipFree(y);

	return 0;
}